#include "hip/hip_runtime.h"
/*
 * STRUMPACK -- STRUctured Matrices PACKage, Copyright (c) 2014, The
 * Regents of the University of California, through Lawrence Berkeley
 * National Laboratory (subject to receipt of any required approvals
 * from the U.S. Dept. of Energy).  All rights reserved.
 *
 * If you have questions about your rights to use or distribute this
 * software, please contact Berkeley Lab's Technology Transfer
 * Department at TTD@lbl.gov.
 *
 * NOTICE. This software is owned by the U.S. Department of Energy. As
 * such, the U.S. Government has been granted for itself and others
 * acting on its behalf a paid-up, nonexclusive, irrevocable,
 * worldwide license in the Software to reproduce, prepare derivative
 * works, and perform publicly and display publicly.  Beginning five
 * (5) years after the date permission to assert copyright is obtained
 * from the U.S. Department of Energy, and subject to any subsequent
 * five (5) year renewals, the U.S. Government is granted for itself
 * and others acting on its behalf a paid-up, nonexclusive,
 * irrevocable, worldwide license in the Software to reproduce,
 * prepare derivative works, distribute copies to the public, perform
 * publicly and display publicly, and to permit others to do so.
 *
 * Developers: Pieter Ghysels, Francois-Henry Rouet, Xiaoye S. Li.
 *             (Lawrence Berkeley National Lab, Computational Research
 *             Division).
 *
 */
#include "CUDAWrapper.hpp"

namespace strumpack {
  namespace gpu {

    template<typename scalar_t> __global__ void
    laswp_kernel(int n, scalar_t* dA, int lddA,
                 int npivots, int* dipiv, int inci) {
      int tid = threadIdx.x + blockDim.x*blockIdx.x;
      if (tid < n) {
        dA += tid * lddA;
        auto A1 = dA;
        for (int i1=0; i1<npivots; i1++) {
          int i2 = dipiv[i1*inci] - 1;
          auto A2 = dA + i2;
          auto temp = *A1;
          *A1 = *A2;
          *A2 = temp;
          A1++;
        }
      }
    }

    template<typename scalar_t> void
    laswp(BLASHandle& handle, DenseMatrix<scalar_t>& dA,
          int k1, int k2, int* dipiv, int inci) {
      if (!dA.rows() || !dA.cols()) return;
      int n = dA.cols(), nt = 256;
      int grid = (n + nt - 1) / nt;
      hipStream_t streamId;
      hipblasGetStream(handle, &streamId);
      laswp_kernel<scalar_t><<<grid, nt, 0, streamId>>>
        (n, dA.data(), dA.ld(), k2-k1+1, dipiv+k1-1, inci);
      gpu_check(hipPeekAtLastError());
    }

    template<typename T>  __global__ void
    laswp_vbatch_kernel(int* dn, T** dA, int* lddA, int** dipiv,
                        int* npivots, unsigned int batchCount) {
      // assume dn = cols, inc = 1
      int x = blockIdx.x * blockDim.x + threadIdx.x,
        f = blockIdx.y * blockDim.y + threadIdx.y;
      if (f >= batchCount) return;
      if (x >= dn[f]) return;
      auto A = dA[f];
      auto P = dipiv[f];
      auto ldA = lddA[f];
      auto npiv = npivots[f];
      A += x * ldA;
      auto A1 = A;
      for (int i=0; i<npiv; i++) {
        auto p = P[i] - 1;
        if (p != i) {
          auto A2 = A + p;
          auto temp = *A1;
          *A1 = *A2;
          *A2 = temp;
        }
        A1++;
      }
    }

    template<typename scalar_t> void
    laswp_fwd_vbatched(BLASHandle& handle, int* dn, int max_n,
                       scalar_t** dA, int* lddA, int** dipiv, int* npivots,
                       unsigned int batchCount) {
      if (max_n <= 0 || !batchCount) return;
      unsigned int nt = 512, ops = 1;
      while (nt > max_n) {
        nt /= 2;
        ops *= 2;
      }
      ops = std::min(ops, batchCount);
      unsigned int nbx = (max_n + nt - 1) / nt,
        nbf = (batchCount + ops - 1) / ops;
      dim3 block(nt, ops), grid(nbx, nbf);
      // assume that nbf < MAX_BLOCKS_Z = 65535, should be ok
      hipStream_t streamId;
      hipblasGetStream(handle, &streamId);
      laswp_vbatch_kernel<<<grid, block, 0, streamId>>>
        (dn, dA, lddA, dipiv, npivots, batchCount);
      gpu_check(hipPeekAtLastError());
    }

    // explicit template instantiations
    template void laswp(BLASHandle&, DenseMatrix<float>&, int, int, int*, int);
    template void laswp(BLASHandle&, DenseMatrix<double>&, int, int, int*, int);
    template void laswp(BLASHandle&, DenseMatrix<std::complex<float>>&, int, int, int*, int);
    template void laswp(BLASHandle&, DenseMatrix<std::complex<double>>&, int, int, int*, int);

    template void laswp_fwd_vbatched(BLASHandle&, int*, int, float**, int*, int**, int*, unsigned int);
    template void laswp_fwd_vbatched(BLASHandle&, int*, int, double**, int*, int**, int*, unsigned int);
    template void laswp_fwd_vbatched(BLASHandle&, int*, int, std::complex<float>**, int*, int**, int*, unsigned int);
    template void laswp_fwd_vbatched(BLASHandle&, int*, int, std::complex<double>**, int*, int**, int*, unsigned int);

  } // end namespace gpu
} // end namespace strumpack
